#include "hip/hip_runtime.h"
//////////////////////////////////////////////
// GPU-BASED PULSE-COUPLED NEURAL NETWORK
// developer : ERIC JANSEN
// e-mail : janseneric[at]gmail[dot]com
// ONLY WORKING UNDER LINUX
//////////////////////////////////////////////

#include "gpupcnnkernel.cuh"

__global__ void gpuProcessS2(cv::gpu::DevMem2D_<float> S,//const cv::gpu::DevMem2D_<float>& S,
		cv::gpu::DevMem2D_<float> S2)//cv::gpu::DevMem2D_<float>& S2)
{
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i > S2.rows || j > S2.cols) return;

  S2.ptr(i)[j] = S.ptr(i)[j]/255.0;
}

__global__ void gpuProcessE(cv::gpu::DevMem2D_<float> E)
{
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i > E.rows || j > E.cols) return;

  E.ptr(i)[j] = 2.0;
}

__global__ void gpuPCNN(cv::gpu::DevMem2Df F,
		cv::gpu::DevMem2Df L,
		cv::gpu::DevMem2Df E,
		cv::gpu::DevMem2Df S2,
		cv::gpu::DevMem2D Sum1,
		cv::gpu::PtrStep Y,
		float dAF,float dAL,float dAE,
		float dVF,float dVE,float dB,
    int R,int C)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= R || j >= C) return;

    float dF,dL,dU,dE;

    dF = F.ptr(i)[j];
    dF = exp(-dAF) * F.ptr(i)[j] + S2.ptr(i)[j] + dVF * Sum1.ptr(i)[j];
    F.ptr(i)[j] = dF;

    dL = L.ptr(i)[j];
    dL = exp(-dAL) * dL + L.ptr(i)[j] * Sum1.ptr(i)[j];
    L.ptr(i)[j] = dL;

    dU = F.ptr(i)[j] * (1 + dB * L.ptr(i)[j]);
    dE = E.ptr(i)[j];
    dE = exp(-dAE) * E.ptr(i)[j] + dVE * Y.ptr(i)[j];
    E.ptr(i)[j] = dE;

    if (dU - dE > 0) Y.ptr(i)[j] = 255;
    else Y.ptr(i)[j] = 0;
}

void callGPUProcessS2(cv::gpu::DevMem2D_<float> S,
    cv::gpu::DevMem2D_<float> S2)
{
  dim3 block(16,16);
  dim3 grid((S2.rows+15)/16,(S2.cols+15)/16);
  gpuProcessS2<<<grid,block>>>(S,S2);
}

void callGPUProcessE(cv::gpu::DevMem2D_<float> E)
{
  dim3 block(16,16);
  dim3 grid((E.rows+15)/16,(E.cols+15)/16);
  gpuProcessE<<<grid,block>>>(E);
}

void callGPUKernel(const cv::gpu::DevMem2Df& F,
	const cv::gpu::DevMem2Df& L,
	const cv::gpu::DevMem2Df& E,
	const cv::gpu::DevMem2Df& S2,
  const cv::gpu::DevMem2D& Sum1,
  cv::gpu::PtrStep Y,
//	const cv::gpu::DevMem2D& Y,
	const float& dAF,const float& dAL,const float& dAE,
	const float& dVF,const float& dVE,const float& dB,
  const int& R,const int& C)
{
  cv::gpu::DevMem2D_<float> gF(F),gL(L),gE(E),gS2(S2);
  cv::gpu::DevMem2D_<unsigned char> gSum1(Sum1);//gY(Y);

	dim3 block(16,16);
	dim3 grid((F.cols+block.x-1)/block.x,(F.rows+block.y-1)/block.y);
	gpuPCNN<<<grid,block>>>(gF,gL,gE,gS2,gSum1,Y,
			dAF,dAL,dAE,dVF,dVE,dB,R,C);
}
